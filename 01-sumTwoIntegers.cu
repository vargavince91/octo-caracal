
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sum(int a, int b, int *c) {
    *c = a + b;
}

int main (void) {

    int a, b, c;

    int *device_c;
    hipMalloc((void**) &device_c, sizeof(int));

    printf("This program calculates the sum of two numbers using GPU.\n");

    printf("a=");
    scanf("%d", &a);
    printf("b=");
    scanf("%d", &b);

    sum<<<1,1>>>(a,b,device_c);

    hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d+%d = %d\n", a, b, c);

    hipFree(device_c);

    return 0;
}
